#include "hip/hip_runtime.h"
﻿

/**
 * EEQAUD Kernel RayTracer
 *
 */

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <time.h>
#include <math.h> 
#include <windows.h>
#include <stdlib.h>
#include <sstream>
#include <string>


#include <glad/gl.h>
#include <GLFW/glfw3.h>


#define N_MAX 1078
#define BMP_HEADER_SIZE 1078

#define texWidth 64
#define texHeight 64

#define IM_WIDTH 640
#define IM_HEIGHT 480
const double M_PI = 3.14159265358979323846;

void read_obj(const char* filename, float** vertex, float** normals, float** color, int* n_vertex, int** faces, int* n_faces);

int write_bmp(const char* filename, int width, int height, unsigned char* rgb);

void readBMP_RGB(char* filename, unsigned char** data_rgb, int* width_rgb, int* height_rgb);
//void processInput(GLFWwindow* window, double* playerX, double* playerY, double* playerAngle, double fovScale, int* map);
void processInput(GLFWwindow* window, double* playerX, double* playerY, double* playerAngle, double fovScale, int mapWidth, int* map);

// Dimensiones de la ventana


GLuint vbo;
struct hipGraphicsResource* cuda_vbo_resource;

GLFWwindow* initOpenGL();
///////////  GPU
__global__ void rayCasting(unsigned char* framebuffer, int screenWidth, int screenHeight, int mapWidth, int mapHeight,
	int* map, unsigned char* textureAtlas, int atlasWidth, int atlasHeight, int texCellWidth, int texCellHeight,
	double posX, double posY, double playerAngle, double fovScale) {

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	double dirX = cos(playerAngle);
	double dirY = sin(playerAngle);
	// Para un FOV de ~60 grados, plane es perpendicular a dir y escalado
	double planeX = -sin(playerAngle) * fovScale;
	double planeY = cos(playerAngle) * fovScale;

	if (x >= screenWidth) return;
	// 1. Calcular raycasting para columna 'x'
	double cameraX = 2.0 * x / double(screenWidth) - 1;
	double rayDirX = dirX + planeX * cameraX;
	double rayDirY = dirY + planeY * cameraX;

	// Posiciones del mapa 
	int mapX = int(posX);
	int mapY = int(posY);
	// Longitudes del rayo hasta cruzar un cuadrado
	double deltaDistX = fabs(1 / rayDirX);
	double deltaDistY = fabs(1 / rayDirY);

	double sideDistX, sideDistY;
	int stepX, stepY;
	int hit = 0, side;

	double perpWallDist;

	if (rayDirX < 0) {
		stepX = -1;
		sideDistX = (posX - mapX) * deltaDistX;
	}
	else {
		stepX = 1;
		sideDistX = (mapX + 1.0 - posX) * deltaDistX;
	}
	if (rayDirY < 0) {
		stepY = -1;
		sideDistY = (posY - mapY) * deltaDistY;
	}
	else {
		stepY = 1;
		sideDistY = (mapY + 1.0 - posY) * deltaDistY;
	}
	//ABSOLUTE CINEMA
	// DDA
	while (hit == 0) {
		if (sideDistX < sideDistY) {
			sideDistX += deltaDistX;
			mapX += stepX;
			side = 0;
		}
		else {
			sideDistY += deltaDistY;
			mapY += stepY;
			side = 1;
		}

		// ❗ Corte seguro si se sale del mapa
		if (mapX < 0 || mapX >= mapWidth || mapY < 0 || mapY >= mapHeight) {
			hit = 1;
			perpWallDist = 1e6; // simula "pared muy lejana"
			break;
		}

		if (map[mapY * mapWidth + mapX] > 0) {
			hit = 1;
		}
	}


	if (side == 0)
		perpWallDist = (sideDistX - deltaDistX);
	else
		perpWallDist = (sideDistY - deltaDistY);

	// Altura de la pared a dibujar en pantalla
	int lineHeight = (int)(screenHeight / perpWallDist);
	int drawStart = -lineHeight / 2 + screenHeight / 2;
	int drawEnd = lineHeight / 2 + screenHeight / 2;

	if (drawStart < 0) drawStart = 0;
	if (drawEnd >= screenHeight) drawEnd = screenHeight - 1;

	// Pintar cielo arriba
	for (int y = 0; y < drawStart; y++) {
		int idx = (y * screenWidth + x) * 3;
		framebuffer[idx + 0] = 135; // R cielo
		framebuffer[idx + 1] = 206; // G
		framebuffer[idx + 2] = 235; // B
	}

	// Pintar suelo abajo
	for (int y = drawEnd + 1; y < screenHeight; y++) {
		int idx = (y * screenWidth + x) * 3;
		framebuffer[idx + 0] = 68; // R suelo
		framebuffer[idx + 1] = 68;
		framebuffer[idx + 2] = 68;
	}

	// === TEXTURIZACIÓN ===

	// Posición exacta del impacto en la pared
	double wallX;
	if (side == 0)
		wallX = posY + perpWallDist * rayDirY;
	else
		wallX = posX + perpWallDist * rayDirX;

	wallX -= floor(wallX); // solo la parte fraccionaria

	int texX = int(wallX * double(texCellWidth));
	if ((side == 0 && rayDirX > 0) || (side == 1 && rayDirY < 0))
		texX = texCellWidth - texX - 1;

	// Para cada pixel vertical de la pared, calcular color y copiar
	for (int y = drawStart; y <= drawEnd; y++) {
		int d = y * 256 - screenHeight * 128 + lineHeight * 128;
		int texY = ((d * texCellHeight) / lineHeight) / 256;

		int atlasX = mapX * texCellWidth + texX;
		int atlasY = mapY * texCellHeight + texY;

		// Evitar accesos fuera de los límites
		if (atlasX < 0 || atlasX >= atlasWidth || atlasY < 0 || atlasY >= atlasHeight)
			continue;

		int texIdx = (atlasY * atlasWidth + atlasX) * 3;
		int fbIdx = (y * screenWidth + x) * 3;

		framebuffer[fbIdx + 0] = textureAtlas[texIdx + 0];
		framebuffer[fbIdx + 1] = textureAtlas[texIdx + 1];
		framebuffer[fbIdx + 2] = textureAtlas[texIdx + 2];
	}
}

//kernel para procesado del mapa
__global__ void drawMapTextureKernel(unsigned char* framebuffer, int fbWidth, int fbHeight,
	int* map, int mapWidth, int mapHeight,
	unsigned char* textureAtlas, int imgWidth, int imgHeight,
	int texCellWidth, int texCellHeight, int cellPixelSize, int channels) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= fbWidth || y >= fbHeight) return;

	int cellX = x / cellPixelSize;
	int cellY = y / cellPixelSize;

	if (cellX >= mapWidth || cellY >= mapHeight) return;

	int pixelInCellX = x % cellPixelSize;
	int pixelInCellY = y % cellPixelSize;

	int cellType = map[cellY * mapWidth + cellX];

	// Textura de la celda dentro del atlas (asumiendo concatenadas horizontalmente)
	int textureOffsetX = cellType * texCellWidth;

	int texX = pixelInCellX * texCellWidth / cellPixelSize;
	int texY = pixelInCellY * texCellHeight / cellPixelSize;

	int imgX = textureOffsetX + texX;
	int imgY = texY;

	int imgIdx = (imgY * imgWidth + imgX) * channels;
	int fbIdx = (y * fbWidth + x) * channels;

	for (int c = 0; c < channels; c++) {
		framebuffer[fbIdx + c] = textureAtlas[imgIdx + c];
	}
}

/*
// crea un buffer y lo registra en cuda --Para despues-- porque hay que instalar mierdas
void createVBO(GLuint* vbo) {
	glGenBuffers(1, vbo);
	glBindBuffer(GL_ARRAY_BUFFER, *vbo);
	glBufferData(GL_ARRAY_BUFFER, WIDTH * HEIGHT * sizeof(float2), nullptr, GL_DYNAMIC_DRAW);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, *vbo, cudaGraphicsMapFlagsWriteDiscard);
}
*/


// Entrada de usuario
double mouseX = 0, mouseY = 0;
bool keys[1024] = { false };

// Callbacks GLFW
void key_callback(GLFWwindow* window, int key, int scancode, int action, int mods) {
	if (key >= 0 && key < 1024) {
		keys[key] = (action != GLFW_RELEASE);
	}
}
void mouse_callback(GLFWwindow* window, double xpos, double ypos) {
	mouseX = xpos;
	mouseY = ypos;
}

GLFWwindow* initOpenGL() {
	//inicializamos glfw para poder crear ventana
	if (!glfwInit()) {
		std::cerr << "GLFW init failed\n";
		return nullptr;
	}
	glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_COMPAT_PROFILE);

	//version 3.3 (la instalada)
	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
	// Creamos ventana
	GLFWwindow* window = glfwCreateWindow(IM_WIDTH, IM_HEIGHT, "OpenGL + CUDA Interop", nullptr, nullptr);
	if (!window) {
		std::cerr << "Failed to create GLFW window\n";
		glfwTerminate();
		return nullptr;
	}

	glfwMakeContextCurrent(window);
	glfwSetKeyCallback(window, key_callback);
	glfwSetCursorPosCallback(window, mouse_callback);
	glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_DISABLED);

	//carga de funciones de openGL modernas
	if (!gladLoadGL(glfwGetProcAddress)) {
		std::cerr << "Error: no se pudo cargar OpenGL con glad\n";
		return nullptr;
	}

	glViewport(0, 0, IM_WIDTH, IM_HEIGHT);
	glPointSize(1.0f); // Tamaño del punto

	return window;
}

__global__ void buildMapFromImage(unsigned char* image, int imgWidth, int imgHeight, int cellSize, int* mapOut) {
	int mapX = blockIdx.x * blockDim.x + threadIdx.x;
	int mapY = blockIdx.y * blockDim.y + threadIdx.y;

	if (mapX >= imgWidth / cellSize || mapY >= imgHeight / cellSize) return;

	int startX = mapX * cellSize;
	int startY = mapY * cellSize;

	// Leer pixel central de la celda
	int pixelIndex = ((startY + cellSize / 2) * imgWidth + (startX + cellSize / 2)) * 3;
	unsigned char r = image[pixelIndex + 0];
	unsigned char g = image[pixelIndex + 1];
	unsigned char b = image[pixelIndex + 2];

	int value;
	double luminosidad = 0.299 * r + 0.587 * g + 0.114 * b;
	// Traduces color a tipo de celda (ejemplo simple)
	if (luminosidad > 200) {
		value = 0; // suelo
	}
	else {
		value = 1; // pared
	}

	mapOut[mapY * (imgWidth / cellSize) + mapX] = value;
}


double moveSpeed = 0.05;

/*
#define mapWidth  24
#define mapHeight 24
*/
void processInput(GLFWwindow* window, double* playerX, double* playerY, double* playerAngle, double fovScale, int mapWidth, int* map) {
	static double lastMouseX;
	static bool firstMouse = true;

	double xpos, ypos;
	glfwGetCursorPos(window, &xpos, &ypos);

	if (firstMouse) {
		lastMouseX = xpos;
		firstMouse = false;
	}

	double mouseDeltaX = xpos - lastMouseX;
	lastMouseX = xpos;

	// Sensibilidad del ratón
	double mouseSensitivity = 0.003;
	*playerAngle += mouseDeltaX * mouseSensitivity;

	// Normalizar ángulo entre 0 y 2PI
	if (*playerAngle < 0) *playerAngle += 2 * M_PI;
	if (*playerAngle >= 2 * M_PI) *playerAngle -= 2 * M_PI;

	// Calcular nueva dirección en base al ángulo actualizado
	double dirX = cos(*playerAngle);
	double dirY = sin(*playerAngle);
	double planeX = -sin(*playerAngle) * fovScale;
	double planeY = cos(*playerAngle) * fovScale;

	double newX = *playerX;
	double newY = *playerY;

	if (glfwGetKey(window, GLFW_KEY_W) == GLFW_PRESS) {
		std::cout << "W PRESSED\n";
		newX += dirX * moveSpeed;
		newY += dirY * moveSpeed;
	}
	if (glfwGetKey(window, GLFW_KEY_S) == GLFW_PRESS) {
		newX -= dirX * moveSpeed;
		newY -= dirY * moveSpeed;
	}
	if (glfwGetKey(window, GLFW_KEY_A) == GLFW_PRESS) {
		newX -= planeX * moveSpeed;
		newY -= planeY * moveSpeed;
	}
	if (glfwGetKey(window, GLFW_KEY_D) == GLFW_PRESS) {
		newX += planeX * moveSpeed;
		newY += planeY * moveSpeed;
	}

	// Verificación de colisión
	int mapIndexX = int(newX);
	int mapIndexY = int(*playerY);
	if (map[mapIndexY * mapWidth + mapIndexX] == 0) {
		*playerX = newX;
	}

	mapIndexX = int(*playerX);
	mapIndexY = int(newY);
	if (map[mapIndexY * mapWidth + mapIndexX] == 0) {
		*playerY = newY;
	}
}


/////// Main

int main()
{
	int ImageWidth = 640;
	int ImageHeight = 480;

	//lectura imagen mapa
	char* filemap = "./mapa.bmp";
	unsigned char* mapa_rgb;
	unsigned char* d_mapa_rgb;
	int w_rgb;
	int h_rgb;
	readBMP_RGB(filemap, &mapa_rgb, &w_rgb, &h_rgb);
	hipMalloc(&d_mapa_rgb, w_rgb * h_rgb * 3);
	hipMemcpy(d_mapa_rgb, mapa_rgb, w_rgb * h_rgb * 3, hipMemcpyHostToDevice);

	int cellSize = 10;
	int mapWidth = w_rgb / cellSize;   // = 24
	int mapHeight = h_rgb / cellSize; // = 24
	int* d_map;

	/*int map[mapWidth * mapHeight] = {
		1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
		1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,6,1,
		1,0,2,0,0,0,0,0,0,0,2,0,0,0,0,0,0,0,0,0,0,0,0,1,
		1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,
		1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,
		1,0,0,0,3,0,0,0,0,0,0,0,5,0,0,0,0,0,0,0,0,0,0,1,
		1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,
		1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1
	};
	*/
	hipMalloc(&d_map, mapWidth * mapHeight * sizeof(int));

	//ventana
	GLFWwindow* window = initOpenGL();
	if (!window) return -1;
	//cargamos texturas
	glViewport(0, 0, ImageWidth, ImageHeight);
	GLuint texID;
	glGenTextures(1, &texID);
	glBindTexture(GL_TEXTURE_2D, texID);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB8, ImageWidth, ImageHeight, 0, GL_RGB, GL_UNSIGNED_BYTE, NULL);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
	glBindTexture(GL_TEXTURE_2D, 0);

	//


	// Posición y dirección del jugador
	double playerX = 6.25;
	double playerY = 6.5;
	double fovScale = 0.66;
	/*double dirX = -1.0;
	double dirY = 0.0;
	double planeX = 0.0;
	double planeY = 0.66;
	*/

	dim3 blockDim1(16, 16);
	dim3 gridDim1((mapWidth + blockDim1.x - 1) / blockDim1.x, (mapHeight + blockDim1.y - 1) / blockDim1.y);

	//esto retorna en d_map un array con los valores y si hay pared o no etc en funcion de la imagen, 
	// o sea es el equivalente a hacer unnmemcpy de map
	buildMapFromImage << <gridDim1, blockDim1 >> > (d_mapa_rgb, w_rgb, h_rgb, cellSize, d_map);
	hipDeviceSynchronize();
	int* h_map = new int[mapWidth * mapHeight];
	hipMemcpy(h_map, d_map, mapWidth * mapHeight * sizeof(int), hipMemcpyDeviceToHost);
	std::cout << "MAPA GENERADO: perro\n";
	for (int y = 0; y < mapHeight; y++) {
		for (int x = 0; x < mapWidth; x++) {
			std::cout << h_map[y * mapWidth + x] << " ";
		}
		std::cout << "\n";
	}
	// Framebuffer
	size_t fbSize = ImageWidth * ImageHeight * 3; //sizeof(unsigned char);
	unsigned char* framebuffer = new unsigned char[ImageWidth * ImageHeight * 3];

	// GPU buffers
	unsigned char* d_framebuffer;
	double playerAngle = 45;
	hipMalloc(&d_framebuffer, fbSize);
	/*int* d_map;
	hipMalloc(&d_map, mapWidth * mapHeight * sizeof(int));
	hipMemcpy(d_map, map, mapWidth * mapHeight * sizeof(int), hipMemcpyHostToDevice);
	*/

	dim3 blockDim(32);
	dim3 gridDim((ImageWidth + blockDim.x - 1) / blockDim.x);

	//bucle de procesado mientras no se cierre la ventana no para
	while (!glfwWindowShouldClose(window)) {

		double dirX = cos(playerAngle);
		double dirY = sin(playerAngle);
		double planeX = -sin(playerAngle) * fovScale;
		double planeY = cos(playerAngle) * fovScale;
		processInput(window, &playerX, &playerY, &playerAngle, fovScale, mapWidth, h_map);
		//copiamos mapa 


		// Configurar kernel


		rayCasting << <gridDim, blockDim >> > (d_framebuffer, ImageWidth, ImageHeight, mapWidth, mapHeight, d_map, d_mapa_rgb, w_rgb, h_rgb, cellSize, cellSize, playerX, playerY, playerAngle, fovScale);//dirX, dirY, planeX, planeY);

		hipDeviceSynchronize();

		hipMemcpy(framebuffer, d_framebuffer, fbSize, hipMemcpyDeviceToHost);
		//le decimos a openGL que use la textura que tiene textID
		glBindTexture(GL_TEXTURE_2D, texID);
		//copiamos buffer de la cpu a la textura activa en la ventana en GPU
		glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, ImageWidth, ImageHeight, GL_RGB, GL_UNSIGNED_BYTE, framebuffer);
		GLenum err = glGetError();
		if (err != GL_NO_ERROR) {
			std::cerr << "OpenGL error: " << err << std::endl;
		}

		//la desactivamos para no modificarla despues sin querer
		glBindTexture(GL_TEXTURE_2D, 0);

		//limpia ventana
		glClear(GL_COLOR_BUFFER_BIT);
		//activamos texturas de openGL y que vamos a usar la textura textID para dibujar 
		glEnable(GL_TEXTURE_2D);
		glBindTexture(GL_TEXTURE_2D, texID);
		//dibujamos un cuadrado que ocupa toda la pantalla y cada vertuce indica la posicion entre -1
		// y 1 que es toda la ventana, cada coordenada indica que parte de la imagen va en ese vertice
		glBegin(GL_QUADS);
		glTexCoord2f(0, 0); glVertex2f(-1, -1);
		glTexCoord2f(1, 0); glVertex2f(1, -1);
		glTexCoord2f(1, 1); glVertex2f(1, 1);
		glTexCoord2f(0, 1); glVertex2f(-1, 1);
		glEnd();
		//desactivamos textura y texturizado para no afectar a otras partes
		glBindTexture(GL_TEXTURE_2D, 0);
		glDisable(GL_TEXTURE_2D);

		// cambiamos buffer actual por el actualizado y dejamos que procese eventos para que siga funcionando
		glfwSwapBuffers(window);
		glfwPollEvents();


	}
	// Guardar imagen
	/*
	unsigned char* rgb = new unsigned char[ImageWidth * ImageHeight * 3];
	for (int i = 0; i < ImageWidth * ImageHeight; ++i) {
		uint32_t color = framebuffer[i];
		rgb[i * 3 + 0] = (color >> 16) & 0xFF; // R
		rgb[i * 3 + 1] = (color >> 8) & 0xFF;  // G
		rgb[i * 3 + 2] = color & 0xFF;         // B
	}

	// Guardar como BMP
	if (!write_bmp("output.bmp", ImageWidth, ImageHeight, rgb)) {
		std::cerr << "Error escribiendo BMP\n";
	}

	delete[] rgb;
	*/
	hipFree(d_framebuffer);
	hipFree(d_map);
	hipFree(d_mapa_rgb);
	delete[] mapa_rgb;
	delete[] framebuffer;

	std::cout << "Imagen renderizada en output.ppm\n";
	return 0;

}

//////// read/write files

struct BMPHeader
{
	char bfType[2];       /* "BM" */
	int bfSize;           /* Size of file in bytes */
	int bfReserved;       /* set to 0 */
	int bfOffBits;        /* Byte offset to actual bitmap data (= 54) */
	int biSize;           /* Size of BITMAPINFOHEADER, in bytes (= 40) */
	int biWidth;          /* Width of image, in pixels */
	int biHeight;         /* Height of images, in pixels */
	short biPlanes;       /* Number of planes in target device (set to 1) */
	short biBitCount;     /* Bits per pixel (24 in this case) */
	int biCompression;    /* Type of compression (0 if no compression) */
	int biSizeImage;      /* Image size, in bytes (0 if no compression) */
	int biXPelsPerMeter;  /* Resolution in pixels/meter of display device */
	int biYPelsPerMeter;  /* Resolution in pixels/meter of display device */
	int biClrUsed;        /* Number of colors in the color table (if 0, use
						  maximum allowed by biBitCount) */
	int biClrImportant;   /* Number of important colors.  If 0, all colors
						  are important */
};

void read_obj(const char* filename, float** vertex, float** normals, float** color, int* n_vertex, int** faces, int* n_faces)
{
	FILE* file = fopen(filename, "r");
	if (file == NULL) {
		printf("Impossible to open the file !\n");
		return;
	}

	int n_faces_l = 0;
	int faces_l[N_MAX * 3];
	int n_vertex_l = 0;
	float vertex_l[N_MAX * 3];
	float colors_l[N_MAX * 3];
	int n_normals_l = 0;
	float normals_l[N_MAX * 3];

	while (1) {

		char lineHeader[128];
		// Lee la primera palabra de la línea
		int res = fscanf(file, "%s", lineHeader);

		if (res == EOF)
			break; // EOF = End Of File, es decir, el final del archivo. Se finaliza el ciclo.

		if (strcmp(lineHeader, "v") == 0) {
			float x, y, z, r, g, b;
			fscanf(file, "%f %f %f %f %f %f \n", &x, &y, &z, &r, &g, &b);

			// guardo vertices 
			vertex_l[n_vertex_l * 3] = x;
			vertex_l[n_vertex_l * 3 + 1] = y;
			vertex_l[n_vertex_l * 3 + 2] = z;

			// guardo colores
			colors_l[n_vertex_l * 3] = r;
			colors_l[n_vertex_l * 3 + 1] = g;
			colors_l[n_vertex_l * 3 + 2] = b;

			n_vertex_l++;
		}

		if (strcmp(lineHeader, "vn") == 0) {
			float x, y, z;
			fscanf(file, "%f %f %f\n", &x, &y, &z);

			// guardo normales 
			normals_l[n_normals_l * 3] = x;
			normals_l[n_normals_l * 3 + 1] = y;
			normals_l[n_normals_l * 3 + 2] = z;

			n_normals_l++;
		}

		if (strcmp(lineHeader, "f") == 0) {
			int i, j, k;
			fscanf(file, "%d//%d %d//%d %d//%d\n", &i, &i, &j, &j, &k, &k);

			// guardo caras (comenzando por 0)
			faces_l[n_faces_l * 3] = i - 1;
			faces_l[n_faces_l * 3 + 1] = j - 1;
			faces_l[n_faces_l * 3 + 2] = k - 1;

			n_faces_l++;
		}
	}

	if (n_normals_l != n_vertex_l) {
		printf("Different number of vertex and normals!!!\n");
		return;
	}

	/// copio datos

	n_vertex[0] = n_vertex_l;
	*vertex = new float[n_vertex_l * 3];
	*normals = new float[n_vertex_l * 3];
	*color = new float[n_vertex_l * 3];

	for (int i_v = 0; i_v < n_vertex_l; i_v++) {
		(*vertex)[i_v * 3] = vertex_l[i_v * 3];
		(*vertex)[i_v * 3 + 1] = vertex_l[i_v * 3 + 1];
		(*vertex)[i_v * 3 + 2] = vertex_l[i_v * 3 + 2];

		(*normals)[i_v * 3] = normals_l[i_v * 3];
		(*normals)[i_v * 3 + 1] = normals_l[i_v * 3 + 1];
		(*normals)[i_v * 3 + 2] = normals_l[i_v * 3 + 2];

		(*color)[i_v * 3] = colors_l[i_v * 3];
		(*color)[i_v * 3 + 1] = colors_l[i_v * 3 + 1];
		(*color)[i_v * 3 + 2] = colors_l[i_v * 3 + 2];
	}

	n_faces[0] = n_faces_l;
	*faces = new int[n_faces_l * 3];

	for (int i_f = 0; i_f < n_faces_l; i_f++) {
		(*faces)[i_f * 3] = faces_l[i_f * 3];
		(*faces)[i_f * 3 + 1] = faces_l[i_f * 3 + 1];
		(*faces)[i_f * 3 + 2] = faces_l[i_f * 3 + 2];
	}
}

int write_bmp(const char* filename, int width, int height, unsigned char* rgb)
{
	int i, j, ipos;
	int bytesPerLine;
	char* line;

	FILE* file;
	struct BMPHeader bmph;

	/* The length of each line must be a multiple of 4 bytes */

	bytesPerLine = (3 * (width + 1) / 4) * 4;

	strcpy(bmph.bfType, "BM");
	bmph.bfOffBits = 54;
	bmph.bfSize = bmph.bfOffBits + bytesPerLine * height;
	bmph.bfReserved = 0;
	bmph.biSize = 40;
	bmph.biWidth = width;
	bmph.biHeight = height;
	bmph.biPlanes = 1;
	bmph.biBitCount = 24;
	bmph.biCompression = 0;
	bmph.biSizeImage = bytesPerLine * height;
	bmph.biXPelsPerMeter = 0;
	bmph.biYPelsPerMeter = 0;
	bmph.biClrUsed = 0;
	bmph.biClrImportant = 0;

	file = fopen(filename, "wb");
	if (file == NULL) return(0);

	fwrite(&bmph.bfType, 2, 1, file);
	fwrite(&bmph.bfSize, 4, 1, file);
	fwrite(&bmph.bfReserved, 4, 1, file);
	fwrite(&bmph.bfOffBits, 4, 1, file);
	fwrite(&bmph.biSize, 4, 1, file);
	fwrite(&bmph.biWidth, 4, 1, file);
	fwrite(&bmph.biHeight, 4, 1, file);
	fwrite(&bmph.biPlanes, 2, 1, file);
	fwrite(&bmph.biBitCount, 2, 1, file);
	fwrite(&bmph.biCompression, 4, 1, file);
	fwrite(&bmph.biSizeImage, 4, 1, file);
	fwrite(&bmph.biXPelsPerMeter, 4, 1, file);
	fwrite(&bmph.biYPelsPerMeter, 4, 1, file);
	fwrite(&bmph.biClrUsed, 4, 1, file);
	fwrite(&bmph.biClrImportant, 4, 1, file);

	line = (char*)malloc(bytesPerLine);

	for (i = height - 1; i >= 0; i--)
	{
		for (j = 0; j < width; j++)
		{
			ipos = 3 * (width * i + j);
			line[3 * j] = rgb[ipos + 2];
			line[3 * j + 1] = rgb[ipos + 1];
			line[3 * j + 2] = rgb[ipos];
		}
		fwrite(line, bytesPerLine, 1, file);
	}

	free(line);
	fclose(file);

	return(1);
}

void readBMP_RGB(char* filename, unsigned char** data_rgb, int* w, int* h)
{
	FILE* f = fopen(filename, "rb");
	char info[54];
	fread(info, sizeof(char), 54, f); // read the 54-byte header

	// extract image height and width from header
	int width = *(int*)&info[18];
	int height = *(int*)&info[22];

	int size = 3 * width * height;

	// allocate 3 bytes per pixel
	*data_rgb = new unsigned char[size];

	// read the rest of the data at once
	fread(*data_rgb, sizeof(unsigned char), size, f);

	// close file
	fclose(f);

	// invert some data
	for (int i = 0; i < size; i += 3)
	{
		unsigned char tmp;
		tmp = (*data_rgb)[i];
		(*data_rgb)[i] = (*data_rgb)[i + 2];
		(*data_rgb)[i + 2] = tmp;
	}

	w[0] = width;
	h[0] = height;
}
